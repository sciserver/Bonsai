#include "hip/hip_runtime.h"
#include "bonsai.h"
#include "support_kernels.cu"
#include <stdio.h>

#if CUDART_VERSION >= 9010
    #include "hip/hip_fp16.h"
#else

#if defined(__CUDACC_RTC__)
#define __CUDA_FP16_DECL__ __host__ __device__
#else /* !__CUDACC_RTC__ */
#define __CUDA_FP16_DECL__ static __device__ __inline__
#endif /* __CUDACC_RTC__ */


typedef struct __align__(4) {
   unsigned int x;
} __half2;
typedef struct __align__(2) {
   unsigned short x;
} __half;

typedef __half2 half2;
__CUDA_FP16_DECL__ __half2 __halves2half2(const __half l, const __half h)
{
   __half2 val;
   asm("{  mov.b32 %0, {%1,%2};}\n"
       : "=r"(val.x) : "h"(l.x), "h"(h.x));
   return val;
}
__CUDA_FP16_DECL__ __half __float2half(const float f)
{
   __half val;
   asm volatile("{  cvt.rn.f16.f32 %0, %1;}\n" : "=h"(val.x) : "f"(f));
   return val;
}
__CUDA_FP16_DECL__ __half __float2half_ru(const float f)
{
   __half val;
   asm volatile("{  cvt.rp.f16.f32 %0, %1;}\n" : "=h"(val.x) : "f"(f));
   return val;
}
__CUDA_FP16_DECL__ __half __float2half_rd(const float f)
{
   __half val;
   asm volatile("{  cvt.rm.f16.f32 %0, %1;}\n" : "=h"(val.x) : "f"(f));
   return val;
}

__CUDA_FP16_DECL__ float __half2float(const __half h)
{
   float val;
   asm volatile("{  cvt.f32.f16 %0, %1;}\n" : "=f"(val) : "h"(h.x));
   return val;
}


__CUDA_FP16_DECL__ float2 __half22float2(const __half2 l)
{
   float hi_float;
   float lo_float;
   asm("{.reg .f16 low,high;\n"
       "  mov.b32 {low,high},%1;\n"
       "  cvt.f32.f16 %0, low;}\n" : "=f"(lo_float) : "r"(l.x));

   asm("{.reg .f16 low,high;\n"
       "  mov.b32 {low,high},%1;\n"
       "  cvt.f32.f16 %0, high;}\n" : "=f"(hi_float) : "r"(l.x));

   return make_float2(lo_float, hi_float);
}
#endif



#include "../profiling/bonsai_timing.h"
PROF_MODULE(compute_propertiesD);

#include "node_specs.h"

static __device__ __forceinline__ void sh_MinMax2(int i, int j, float3 *r_min, float3 *r_max, volatile float3 *sh_rmin, volatile  float3 *sh_rmax)
{
  sh_rmin[i].x  = (*r_min).x = fminf((*r_min).x, sh_rmin[j].x);
  sh_rmin[i].y  = (*r_min).y = fminf((*r_min).y, sh_rmin[j].y);
  sh_rmin[i].z  = (*r_min).z = fminf((*r_min).z, sh_rmin[j].z);
  sh_rmax[i].x  = (*r_max).x = fmaxf((*r_max).x, sh_rmax[j].x);
  sh_rmax[i].y  = (*r_max).y = fmaxf((*r_max).y, sh_rmax[j].y);
  sh_rmax[i].z  = (*r_max).z = fmaxf((*r_max).z, sh_rmax[j].z);
}

//////////////////////////////
//////////////////////////////
//////////////////////////////

//Helper functions for leaf-nodes
static __device__ void compute_monopole(double &mass, double &posx,
                                 double &posy, double &posz,
                                 float4 pos)
{
  mass += pos.w;
  posx += pos.w*pos.x;
  posy += pos.w*pos.y;
  posz += pos.w*pos.z;
}

static __device__ void compute_quadropole(double &oct_q11, double &oct_q22, double &oct_q33,
                                   double &oct_q12, double &oct_q13, double &oct_q23,
                                   float4 pos)
{
  oct_q11 += pos.w * pos.x*pos.x;
  oct_q22 += pos.w * pos.y*pos.y;
  oct_q33 += pos.w * pos.z*pos.z;
  oct_q12 += pos.w * pos.x*pos.y;
  oct_q13 += pos.w * pos.y*pos.z;
  oct_q23 += pos.w * pos.z*pos.x;
}

static __device__ void compute_bounds(float3 &r_min, float3 &r_max,
                               float4 pos)
{
  r_min.x = fminf(r_min.x, pos.x);
  r_min.y = fminf(r_min.y, pos.y);
  r_min.z = fminf(r_min.z, pos.z);

  r_max.x = fmaxf(r_max.x, pos.x);
  r_max.y = fmaxf(r_max.y, pos.y);
  r_max.z = fmaxf(r_max.z, pos.z);
}

//Non-leaf node helper functions
static __device__ void compute_monopole_node(double &mass, double &posx,
                                 double &posy, double &posz,
                                 double4  pos)
{
  mass += pos.w;
  posx += pos.w*pos.x;
  posy += pos.w*pos.y;
  posz += pos.w*pos.z;
}


static __device__ void compute_quadropole_node(double &oct_q11, double &oct_q22, double &oct_q33,
                                        double &oct_q12, double &oct_q13, double &oct_q23,
                                        double4 Q0, double4 Q1)
{
  oct_q11 += Q0.x;
  oct_q22 += Q0.y;
  oct_q33 += Q0.z;
  oct_q12 += Q1.x;
  oct_q13 += Q1.y;
  oct_q23 += Q1.z;
}

static __device__ void compute_bounds_node(float3 &r_min, float3 &r_max,
                                    float4 node_min, float4 node_max)
{
  r_min.x = fminf(r_min.x, node_min.x);
  r_min.y = fminf(r_min.y, node_min.y);
  r_min.z = fminf(r_min.z, node_min.z);

  r_max.x = fmaxf(r_max.x, node_max.x);
  r_max.y = fmaxf(r_max.y, node_max.y);
  r_max.z = fmaxf(r_max.z, node_max.z);
}

KERNEL_DECLARE(compute_leaf_sph)(
                                 const int       n_leafs,
                                 uint           *leafsIdxs,
                                 uint2          *node_bodies,
                                 real4          *body_pos,
                                 double4        *multipole,
                                 real4          *nodeLowerBounds,
                                 real4          *nodeUpperBounds,
                                 real4          *body_vel,
                                 ulonglong1     *body_id,
                                 real           *body_h, //TODO REMOVE, not used anymore
                                 const float     h_min,  //TODO REMOVE, not used anymore
                                 const float2   *body_dens) {

  CUXTIMER("compute_sph_leaf");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint id  = bid * blockDim.x + tid;

  if (id >= n_leafs) return;


  //Since nodes are intermixed with non-leafs in the node_bodies array
  //we get a leaf-id from the leafsIdxs array
  int nodeID = leafsIdxs[id];

  const uint2 bij          =  node_bodies[nodeID];
  const uint firstChild    =  bij.x & ILEVELMASK;
  const uint lastChild     =  bij.y;  //TODO maybe have to increase it by 1

  //Variables holding properties and intermediate answers
  float4 p;

  double mass, posx, posy, posz;
  mass = posx = posy = posz = 0.0;

  double oct_q11, oct_q22, oct_q33;
  double oct_q12, oct_q13, oct_q23;
  oct_q11 = oct_q22 = oct_q33 = 0.0;
  oct_q12 = oct_q13 = oct_q23 = 0.0;

  float3 r_min = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_max = make_float3(-1e10f, -1e10f, -1e10f);

  //Loop over the children=>particles=>bodys
  //unroll increases register usage #pragma unroll 16
  float maxEps  = -100.0f;
  float maxSmth = -100.0f;
  for(int i=firstChild; i < lastChild; i++)
  {
    p      = body_pos[i];

    compute_monopole(mass, posx, posy, posz, p);
    compute_quadropole(oct_q11, oct_q22, oct_q33, oct_q12, oct_q13, oct_q23, p);
    //    compute_bounds(r_min, r_max, p);

    maxSmth = fmaxf(body_dens[i].y, maxSmth);

    //Change the particle into a box and use that as the boundaries
    float  smth  = body_dens[i].y*SPH_KERNEL_SIZE;
    compute_bounds_node(r_min, r_max,
                        make_float4(p.x-smth, p.y-smth, p.z-smth, 0),
                        make_float4(p.x+smth, p.y+smth, p.z+smth, 0));

  }

  double4 mon = {posx, posy, posz, mass};

  double im = 1.0/mon.w;
  if(mon.w == 0) im = 0;        //Allow tracer/massless particles
  mon.x *= im;
  mon.y *= im;
  mon.z *= im;


  double4 Q0, Q1;
  Q0 = make_double4(oct_q11, oct_q22, oct_q33, maxEps); //Store max softening
  Q1 = make_double4(oct_q12, oct_q13, oct_q23, 0.0f);

  //Store the leaf properties
  multipole[3*nodeID + 0] = mon;       //Monopole
  multipole[3*nodeID + 1] = Q0;        //Quadropole
  multipole[3*nodeID + 2] = Q1;        //Quadropole

  //Store the node boundaries
  nodeLowerBounds[nodeID] = make_float4(r_min.x, r_min.y, r_min.z, maxSmth); //4th parameter holds the maximum smoothing range of underlying particles
  nodeUpperBounds[nodeID] = make_float4(r_max.x, r_max.y, r_max.z, 1.0f);    //4th parameter is set to 1 to indicate this is a leaf

  return;
}


KERNEL_DECLARE(compute_leaf)
                            (const int         n_leafs,
                                   uint       *leafsIdxs,
                                   uint2      *node_bodies,
                                   real4      *body_pos,
                                   double4    *multipole,
                                   real4      *nodeLowerBounds,
                                   real4      *nodeUpperBounds,
                                   real4      *body_vel,
                                   ulonglong1 *body_id,
                                   real       *body_h,
			                 const float       h_min,
			                 const float2     *body_dens) {

  CUXTIMER("compute_leaf");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint id  = bid * blockDim.x + tid;

  //Set the shared memory for these threads and exit the thread
  if (id >= n_leafs) return;


  //Since nodes are intermixes with non-leafs in the node_bodies array
  //we get a leaf-id from the leafsIdxs array
  int nodeID = leafsIdxs[id];

  const uint2 bij          =  node_bodies[nodeID];
  const uint firstChild    =  bij.x & ILEVELMASK;
  const uint lastChild     =  bij.y;  //TODO maybe have to increase it by 1

  //Variables holding properties and intermediate answers
  float4 p;

  double mass, posx, posy, posz;
  mass = posx = posy = posz = 0.0;

  double oct_q11, oct_q22, oct_q33;
  double oct_q12, oct_q13, oct_q23;

  oct_q11 = oct_q22 = oct_q33 = 0.0;
  oct_q12 = oct_q13 = oct_q23 = 0.0;

  float3  r_min = make_float3(+1e10f, +1e10f, +1e10f);
  float3  r_max = make_float3(-1e10f, -1e10f, -1e10f);

  float3 r_minSPH = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_maxSPH = make_float3(-1e10f, -1e10f, -1e10f);

  //Loop over the children=>particles=>bodys
  //unroll increases register usage #pragma unroll 16
  float maxEps  = -100.0f;
  float maxSmth = -100.0f;
  int count=0;
  for(int i=firstChild; i < lastChild; i++)
  {
    p      = body_pos[i];
    maxEps = fmaxf(body_vel[i].w, maxEps);      //Determine the max softening within this leaf
    count++;

    compute_monopole(mass, posx, posy, posz, p);
    compute_quadropole(oct_q11, oct_q22, oct_q33, oct_q12, oct_q13, oct_q23, p);
    compute_bounds(r_min, r_max, p);

//    maxSmth = fmaxf(body_dens[i].y, maxSmth);
    //Change the particle into a box and use that as the boundaries
    float  smth  = body_dens[i].y*SPH_KERNEL_SIZE;
    compute_bounds_node(r_minSPH, r_maxSPH,
                        make_float4(p.x-smth, p.y-smth, p.z-smth, 0),
                        make_float4(p.x+smth, p.y+smth, p.z+smth, 0));
  }

  //SPH max smoothing value for the cell
  //Uses the box-extends which makes the box tighter and hence more efficient. ( for example if (boxEdge-particle.pos) > particle-smoothing)

  //Compute the max smoothing range for this cell by determining the max distance between the tightbox and the smoothedbox
  maxSmth     = fmaxf(fmaxf(fmaxf(fabs(r_min.x-r_minSPH.x), fabs(r_max.x-r_maxSPH.x)),
                            fmaxf(fabs(r_min.y-r_minSPH.y), fabs(r_max.y-r_maxSPH.y))),
                            fmaxf(fabs(r_min.z-r_minSPH.z), fabs(r_max.z-r_maxSPH.z)));


  double4 mon = {posx, posy, posz, mass};

  double im = 1.0/mon.w;
  if(mon.w == 0) im = 0;        //Allow tracer/massless particles
  mon.x *= im;
  mon.y *= im;
  mon.z *= im;

  double4 Q0, Q1;
  Q0 = make_double4(oct_q11, oct_q22, oct_q33, maxEps); //Store max softening
  Q1 = make_double4(oct_q12, oct_q13, oct_q23, 0.0f);

  //Store the leaf properties
  multipole[3*nodeID + 0] = mon;       //Monopole
  multipole[3*nodeID + 1] = Q0;        //Quadropole
  multipole[3*nodeID + 2] = Q1;        //Quadropole

  //Store the node boundaries
  nodeLowerBounds[nodeID] = make_float4(r_min.x, r_min.y, r_min.z, maxSmth); //4th parameter holds the maximum smoothing range of underlying particles
  nodeUpperBounds[nodeID] = make_float4(r_max.x, r_max.y, r_max.z, 1.0f);    //4th parameter is set to 1 to indicate this is a leaf

//Compute the box physical center and size
//  float3 boxCenter,boxSize;
//  boxCenter.x   = make_float3(0.5*(r_min.x + r_max.x),
//                              0.5*(r_min.y + r_max.y),
//                              0.5*(r_min.z + r_max.z));
//  boxSize       = make_float3(fmaxf(fabs(boxCenter.x-r_min.x), fabs(boxCenter.x-r_max.x)),
//                              fmaxf(fabs(boxCenter.y-r_min.y), fabs(boxCenter.y-r_max.y)),
//                              fmaxf(fabs(boxCenter.z-r_min.z), fabs(boxCenter.z-r_max.z)));
//Do the same thing for the smoothed size





#if 1
    const float3 len = make_float3(r_max.x-r_min.x, r_max.y-r_min.y, r_max.z-r_min.z);
    const float  vol = cbrtf(len.x*len.y*len.z);
    float hp  = 0;
    if (vol > 0.0f)
    {
      const float nd  = float(lastChild - firstChild) / vol;
      hp  = cbrtf(42.0f / nd);
    }
    hp = max(hp, h_min);
    for(int i=firstChild; i < lastChild; i++)
      if(body_h[i] < 0)
        body_h[i] = hp;
#endif

  return;
}


//Function goes level by level (starting from deepest) and computes
//the properties of the non-leaf nodes
KERNEL_DECLARE(compute_non_leaf)(const int curLevel,         //Level for which we calc
                                 uint  *leafsIdxs,           //Conversion of ids
                                 uint  *node_level_list,     //Contains the start nodes of each lvl
                                 uint  *n_children,          //Reference from node to first child and number of childs
                                 double4 *multipole,
                                 real4 *nodeLowerBounds,
                                 real4 *nodeUpperBounds){

  CUXTIMER("compute_non_leaf");
  const int bid =  blockIdx.y *  gridDim.x +  blockIdx.x;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;

  const int idx = bid * (blockDim.x * blockDim.y) + tid;

  const int endNode   = node_level_list[curLevel];
  const int startNode = node_level_list[curLevel-1];


  if(idx >= (endNode-startNode))     return;

  const int nodeID = leafsIdxs[idx + startNode];

  //Get the children info
  const uint firstChild = n_children[nodeID] & 0x0FFFFFFF;                  //TODO make this name/define?
  const uint nChildren  = ((n_children[nodeID]  & 0xF0000000) >> 28); //TODO make this name/define?

  //Variables
  double mass, posx, posy, posz;
  mass = posx = posy = posz = 0.0;

  double oct_q11, oct_q22, oct_q33;
  double oct_q12, oct_q13, oct_q23;

  oct_q11 = oct_q22 = oct_q33 = 0.0;
  oct_q12 = oct_q13 = oct_q23 = 0.0;

  float3 r_min, r_max;
  r_min = make_float3(+1e10f, +1e10f, +1e10f);
  r_max = make_float3(-1e10f, -1e10f, -1e10f);

  float3 r_minSPH = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_maxSPH = make_float3(-1e10f, -1e10f, -1e10f);

  //Process the children (1 to 8)
  float maxEps  = -100.0f;
  float maxSmth = -100.0f;
  for(int i=firstChild; i < firstChild+nChildren; i++)
  {
    //Gogo process this data!
    double4 tmon = multipole[3*i + 0];

    maxEps  = max(multipole[3*i + 1].w, maxEps);


    compute_monopole_node(mass, posx, posy, posz, tmon);
    compute_quadropole_node(oct_q11, oct_q22, oct_q33, oct_q12, oct_q13, oct_q23,
                            multipole[3*i + 1], multipole[3*i + 2]);

    compute_bounds_node(r_min, r_max, nodeLowerBounds[i], nodeUpperBounds[i]);
    compute_bounds_node(r_minSPH, r_maxSPH,
                        make_float4(nodeLowerBounds[i].x-nodeLowerBounds[i].w, nodeLowerBounds[i].y-nodeLowerBounds[i].w, nodeLowerBounds[i].z-nodeLowerBounds[i].w, 0),
                        make_float4(nodeUpperBounds[i].x+nodeLowerBounds[i].w, nodeUpperBounds[i].y+nodeLowerBounds[i].w, nodeUpperBounds[i].z+nodeLowerBounds[i].w, 0));
  }

  maxSmth     = fmaxf(fmaxf(fmaxf(fabs(r_min.x-r_minSPH.x), fabs(r_max.x-r_maxSPH.x)),
                            fmaxf(fabs(r_min.y-r_minSPH.y), fabs(r_max.y-r_maxSPH.y))),
                            fmaxf(fabs(r_min.z-r_minSPH.z), fabs(r_max.z-r_maxSPH.z)));


  //Save the bounds
  nodeLowerBounds[nodeID] = make_float4(r_min.x, r_min.y, r_min.z, maxSmth);
  nodeUpperBounds[nodeID] = make_float4(r_max.x, r_max.y, r_max.z, 0.0f); //4th is set to 0 to indicate a non-leaf

  //Regularize and store the results
  double4 mon = {posx, posy, posz, mass};
  double im = 1.0/mon.w;
  if(mon.w == 0) im = 0; //Allow tracer/massless particles

  mon.x *= im;
  mon.y *= im;
  mon.z *= im;

  double4 Q0, Q1;
  Q0 = make_double4(oct_q11, oct_q22, oct_q33, maxEps); //store max Eps
  Q1 = make_double4(oct_q12, oct_q13, oct_q23, 0.0f);

  multipole[3*nodeID + 0] = mon;        //Monopole
  multipole[3*nodeID + 1] = Q0;         //Quadropole1
  multipole[3*nodeID + 2] = Q1;         //Quadropole2

  return;
}
KERNEL_DECLARE(compute_scaling)(const int      node_count,
                                      double4 *multipole,
                                      real4   *nodeLowerBounds,
                                      real4   *nodeUpperBounds,
                                      uint    *n_children,
                                      real4   *multipoleF,
                                      float    theta,
                                      real4   *boxSizeInfo,
                                      real4   *boxCenterInfo,
                                      uint2   *node_bodies){

  CUXTIMER("compute_scaling");
  const int bid =  blockIdx.y *  gridDim.x +  blockIdx.x;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;

  const int idx = bid * (blockDim.x * blockDim.y) + tid;

  if(idx >= node_count)     return;

  double4 monD, Q0, Q1;

  monD = multipole[3*idx + 0];        //Monopole
  Q0   = multipole[3*idx + 1];        //Quadropole1
  Q1   = multipole[3*idx + 2];        //Quadropole2

  //Scale the quadropole
  double im = 1.0 / monD.w;
  if(monD.w == 0) im = 0;               //Allow tracer/massless particles
  Q0.x = Q0.x*im - monD.x*monD.x;
  Q0.y = Q0.y*im - monD.y*monD.y;
  Q0.z = Q0.z*im - monD.z*monD.z;
  Q1.x = Q1.x*im - monD.x*monD.y;
  Q1.y = Q1.y*im - monD.y*monD.z;
  Q1.z = Q1.z*im - monD.x*monD.z;

  //Switch the y and z parameter
  double temp = Q1.y;
  Q1.y = Q1.z; Q1.z = temp;

  //Convert the doubles to floats
  float4 mon            = make_float4(monD.x, monD.y, monD.z, monD.w);
  multipoleF[3*idx + 0] = mon;
  multipoleF[3*idx + 1] = make_float4(Q0.x, Q0.y, Q0.z, Q0.w);        //Quadropole1
  multipoleF[3*idx + 2] = make_float4(Q1.x, Q1.y, Q1.z, Q1.w);        //Quadropole2

  float4 r_min, r_max;
  r_min = nodeLowerBounds[idx];
  r_max = nodeUpperBounds[idx];

  //Compute center and size of the box

  float3 boxCenter;
  boxCenter.x = 0.5*(r_min.x + r_max.x);
  boxCenter.y = 0.5*(r_min.y + r_max.y);
  boxCenter.z = 0.5*(r_min.z + r_max.z);

  float3 boxSize = make_float3(fmaxf(fabs(boxCenter.x-r_min.x), fabs(boxCenter.x-r_max.x)),
                               fmaxf(fabs(boxCenter.y-r_min.y), fabs(boxCenter.y-r_max.y)),
                               fmaxf(fabs(boxCenter.z-r_min.z), fabs(boxCenter.z-r_max.z)));

  //Calculate distance between center of the box and the center of mass
  float3 s3     = make_float3((boxCenter.x - mon.x), (boxCenter.y - mon.y), (boxCenter.z - mon.z));
  double s      = sqrt((s3.x*s3.x) + (s3.y*s3.y) + (s3.z*s3.z));

  //If mass-less particles form a node, the s would be huge in opening angle, make it 0
  if(fabs(mon.w) < 1e-10) s = 0;

  //Length of the box, note times 2 since we only computed half the distance before
  float l = 2*fmaxf(boxSize.x, fmaxf(boxSize.y, boxSize.z));

  //Store the box size and opening criteria
  boxSizeInfo[idx].x = boxSize.x;
  boxSizeInfo[idx].y = boxSize.y;
  boxSizeInfo[idx].z = boxSize.z;
  boxSizeInfo[idx].w = __int_as_float(n_children[idx]);

#if 1
  boxCenterInfo[idx].x = boxCenter.x;
  boxCenterInfo[idx].y = boxCenter.y;
  boxCenterInfo[idx].z = boxCenter.z;
#else /* added by egaburov, see dev_approximate_gravity_warp.cu for matching code*/
  boxCenterInfo[idx].x = mon.x;
  boxCenterInfo[idx].y = mon.y;
  boxCenterInfo[idx].z = mon.z;
#endif

  uint2 bij     = node_bodies[idx];
  uint pfirst   = bij.x & ILEVELMASK;
  uint nchild   = bij.y - pfirst;

  //Change the indirections of the leaf nodes so
  //they point to the particle data
  bool leaf = (r_max.w > 0);
  if(leaf)
  {
    pfirst             = pfirst | ((nchild-1) << LEAFBIT);
    boxSizeInfo[idx].w = __int_as_float(pfirst);
  }



  //Extra check, shouldn't be necessary, probably it is otherwise the test for leaf can fail
  //So it IS important Otherwise 0.0 < 0 can fail, now it will be: -1e-12 < 0 
  if(l < 0.000001)
    l = 0.000001;

  #ifdef IMPBH
    float cellOp = (l/theta) + s;
  #else
    //Minimum distance method
    float cellOp = (l/theta); 
  #endif
    
  cellOp = cellOp*cellOp;


  //If this is (leaf)node with only 1 particle then we change 
  //the opening criteria to a large number to force that the 
  //leaf will be opened and the particle data is used 
  //instead of an approximation.
  //This because sometimes (mass*pos)*(1.0/mass) != pos
  //even in full double precision
  if(nchild == 1)
  {
    cellOp = 10e10; //Force this node to be opened
  }

  if(r_max.w > 0)
  {
    cellOp = -cellOp;       //This is a leaf node
  }

  boxCenterInfo[idx].w = cellOp;


  bool doSPH = true;
  if(doSPH)
  {
      //Make sure that value is non-zero
      if(r_min.w < 0.000001) r_min.w = 0.000001;
      if (r_max.w > 0){
          boxCenterInfo[idx].w    = -(r_min.w*r_min.w);
      }
      else {
          boxCenterInfo[idx].w    =  (r_min.w*r_min.w);
      }
  }

#if 1
    //Use half precision to store both gravity and sph criteria in a single fp32 value
    __half sph_opening;
    __half tree_opening;

    //Round up and down to make sure we stay on the save side during the actual comparisons
    if (r_max.w > 0) {
      sph_opening  = __float2half_rd(-(r_min.w*r_min.w));
      tree_opening = __float2half_rd(cellOp);
    } else {
      sph_opening  = __float2half_ru(  r_min.w*r_min.w);
      tree_opening = __float2half_ru(cellOp);
    }

    half2 opening = __halves2half2(tree_opening, sph_opening);
    *((half2*)&boxCenterInfo[idx].w) = opening;
#endif

  return;
}


//Compute the properties for the groups
KERNEL_DECLARE(gpu_setPHGroupData)(const int n_groups,
                                          const int n_particles,   
                                          real4 *bodies_pos,
                                          int2  *group_list,                                                
                                          real4 *groupCenterInfo,
                                          real4 *groupSizeInfo){
  CUXTIMER("setPHGroupData");
  const int bid =  blockIdx.y *  gridDim.x +  blockIdx.x;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;

  if(bid >= n_groups)     return;

  //We set a default amount of shared memory to make sure that the reduction has enough memory
  //independent of the number of threads that is actually launched.
  #if NCRIT > 64
      #error "Fatal, NCRIT > 64 increase shared memory allocation below"
  #endif

  //Do a reduction on the particles assigned to this group
  const int NCRIT_TEMP = 64;
  volatile __shared__ float3 shmem[2*NCRIT_TEMP];
  volatile float3 *sh_rmin = (float3*)&shmem [ 0];
  volatile float3 *sh_rmax = (float3*)&shmem[NCRIT_TEMP];

  float3 r_min = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_max = make_float3(-1e10f, -1e10f, -1e10f);

  int start = group_list[bid].x;
  int end   = group_list[bid].y;
  
  int partIdx = start + threadIdx.x;

  //Set the shared memory with the data
  if (partIdx >= end)
  {
    sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
    sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;
  }
  else
  {
    sh_rmin[tid].x = r_min.x = bodies_pos[partIdx].x; sh_rmin[tid].y = r_min.y = bodies_pos[partIdx].y; sh_rmin[tid].z = r_min.z = bodies_pos[partIdx].z;
    sh_rmax[tid].x = r_max.x = bodies_pos[partIdx].x; sh_rmax[tid].y = r_max.y = bodies_pos[partIdx].y; sh_rmax[tid].z = r_max.z = bodies_pos[partIdx].z;
  }


  __syncthreads();
  // do reduction in shared mem  
  if(blockDim.x >= 512) if (tid < 256) {sh_MinMax2(tid, tid + 256, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 256) if (tid < 128) {sh_MinMax2(tid, tid + 128, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 128) if (tid < 64)  {sh_MinMax2(tid, tid + 64,  &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();

  if(blockDim.x >= 64) if (tid < 32)  {sh_MinMax2(tid, tid + 32, &r_min, &r_max, sh_rmin, sh_rmax); }
  if(blockDim.x >= 32) if (tid < 16) { sh_MinMax2(tid, tid + 16, &r_min, &r_max, sh_rmin, sh_rmax); }

  if(tid < 8)
  {
    sh_MinMax2(tid, tid +  8, &r_min, &r_max, sh_rmin, sh_rmax);
    sh_MinMax2(tid, tid +  4, &r_min, &r_max, sh_rmin, sh_rmax);
    sh_MinMax2(tid, tid +  2, &r_min, &r_max, sh_rmin, sh_rmax);
    sh_MinMax2(tid, tid +  1, &r_min, &r_max, sh_rmin, sh_rmax);
  }
  
  // write result for this block to global mem
  if (tid == 0)
  {

    //Compute the group center and size
    float3 grpCenter;
    grpCenter.x = 0.5*(r_min.x + r_max.x);
    grpCenter.y = 0.5*(r_min.y + r_max.y);
    grpCenter.z = 0.5*(r_min.z + r_max.z);

    float3 grpSize = make_float3(fmaxf(fabs(grpCenter.x-r_min.x), fabs(grpCenter.x-r_max.x)),
                                 fmaxf(fabs(grpCenter.y-r_min.y), fabs(grpCenter.y-r_max.y)),
                                 fmaxf(fabs(grpCenter.z-r_min.z), fabs(grpCenter.z-r_max.z)));

    //Store the box size and opening criteria
    groupSizeInfo[bid].x = grpSize.x;
    groupSizeInfo[bid].y = grpSize.y;
    groupSizeInfo[bid].z = grpSize.z;

    int nchild             = end-start;
    start                  = start | (nchild-1) << CRITBIT;
    groupSizeInfo[bid].w   = __int_as_float(start);  

    float l = max(grpSize.x, max(grpSize.y, grpSize.z));

    groupCenterInfo[bid].x = grpCenter.x;
    groupCenterInfo[bid].y = grpCenter.y;
    groupCenterInfo[bid].z = grpCenter.z;

    //Test stats for physical group size
    groupCenterInfo[bid].w = l;

  } //end tid == 0
}//end copyNode2grp



#if 0
//Compute the properties for the groups
KERNEL_DECLARE(gpu_setPHGroupDataGetKey)(const int n_groups,
                                          const int n_particles,
                                          real4 *bodies_pos,
                                          int2  *group_list,
                                          real4 *groupCenterInfo,
                                          real4 *groupSizeInfo,
                                          uint4  *body_key,
                                          float4 corner){
  CUXTIMER("setPHGroupDataGetKey");
  const int bid =  blockIdx.y *  gridDim.x +  blockIdx.x;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;

  if(bid >= n_groups)     return;

  //Do a reduction on the particles assigned to this group

  volatile __shared__ float3 shmem[2*NCRIT];
  volatile float3 *sh_rmin = (float3*)&shmem [ 0];
  volatile float3 *sh_rmax = (float3*)&shmem[NCRIT];

  float3 r_min = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_max = make_float3(-1e10f, -1e10f, -1e10f);

  int start = group_list[bid].x;
  int end   = group_list[bid].y;

  int partIdx = start + threadIdx.x;

  //Set the shared memory with the data
  if (partIdx >= end)
  {
    sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
    sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;
  }
  else
  {
    sh_rmin[tid].x = r_min.x = bodies_pos[partIdx].x; sh_rmin[tid].y = r_min.y = bodies_pos[partIdx].y; sh_rmin[tid].z = r_min.z = bodies_pos[partIdx].z;
    sh_rmax[tid].x = r_max.x = bodies_pos[partIdx].x; sh_rmax[tid].y = r_max.y = bodies_pos[partIdx].y; sh_rmax[tid].z = r_max.z = bodies_pos[partIdx].z;
  }


  __syncthreads();
  // do reduction in shared mem
  if(blockDim.x >= 512) if (tid < 256) {sh_MinMax2(tid, tid + 256, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 256) if (tid < 128) {sh_MinMax2(tid, tid + 128, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 128) if (tid < 64)  {sh_MinMax2(tid, tid + 64,  &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();

  if(blockDim.x >= 64) if (tid < 32)  {sh_MinMax2(tid, tid + 32, &r_min, &r_max, sh_rmin, sh_rmax); }
  if(blockDim.x >= 32) if (tid < 16) { sh_MinMax2(tid, tid + 16, &r_min, &r_max, sh_rmin, sh_rmax); }

  if(tid < 8)
  {
    sh_MinMax2(tid, tid +  8, &r_min, &r_max, sh_rmin, sh_rmax);
    sh_MinMax2(tid, tid +  4, &r_min, &r_max, sh_rmin, sh_rmax);
    sh_MinMax2(tid, tid +  2, &r_min, &r_max, sh_rmin, sh_rmax);
    sh_MinMax2(tid, tid +  1, &r_min, &r_max, sh_rmin, sh_rmax);
  }

  // write result for this block to global mem
  if (tid == 0)
  {

    //Compute the group center and size
    float3 grpCenter;
    grpCenter.x = 0.5*(r_min.x + r_max.x);
    grpCenter.y = 0.5*(r_min.y + r_max.y);
    grpCenter.z = 0.5*(r_min.z + r_max.z);

    float3 grpSize = make_float3(fmaxf(fabs(grpCenter.x-r_min.x), fabs(grpCenter.x-r_max.x)),
                                 fmaxf(fabs(grpCenter.y-r_min.y), fabs(grpCenter.y-r_max.y)),
                                 fmaxf(fabs(grpCenter.z-r_min.z), fabs(grpCenter.z-r_max.z)));

    //Store the box size and opening criteria
    groupSizeInfo[bid].x = grpSize.x;
    groupSizeInfo[bid].y = grpSize.y;
    groupSizeInfo[bid].z = grpSize.z;

    real4 pos = bodies_pos[start];

    int nchild             = end-start;
    start                  = start | (nchild-1) << CRITBIT;
    groupSizeInfo[bid].w   = __int_as_float(start);

    float l = max(grpSize.x, max(grpSize.y, grpSize.z));

    groupCenterInfo[bid].x = grpCenter.x;
    groupCenterInfo[bid].y = grpCenter.y;
    groupCenterInfo[bid].z = grpCenter.z;

    //Test stats for physical group size
    groupCenterInfo[bid].w = l;

    int4 crd;

    real domain_fac = corner.w;

    #ifndef EXACT_KEY
       crd.x = (int)roundf(__fdividef((pos.x - corner.x), domain_fac));
       crd.y = (int)roundf(__fdividef((pos.y - corner.y) , domain_fac));
       crd.z = (int)roundf(__fdividef((pos.z - corner.z) , domain_fac));
    #else
       crd.x = (int)((pos.x - corner.x) / domain_fac);
       crd.y = (int)((pos.y - corner.y) / domain_fac);
       crd.z = (int)((pos.z - corner.z) / domain_fac);
    #endif

    body_key[bid] = get_key(crd);

  } //end tid == 0
}//end copyNode2grp

//Compute the key for the groups
KERNEL_DECLARE(gpu_setPHGroupDataGetKey2)(const int n_groups,
                                      real4 *bodies_pos,
                                      int2  *group_list,
                                      uint4  *body_key,
                                      float4 corner){
  CUXTIMER("setPHGroupDataGetKey2");
  const int bid =  blockIdx.y *  gridDim.x +  blockIdx.x;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int idx = bid * (blockDim.x * blockDim.y) + tid;

  if(idx >= n_groups)     return;


  int start = group_list[idx].x;
  real4 pos = bodies_pos[start];

//  int end   = group_list[idx].y-1;
//  real4 pos = bodies_pos[end];

//  int end   = group_list[idx].y-1;
//  int start = group_list[idx].x;
//  start     = (end+start) / 2;
//  real4 pos = bodies_pos[start];


  int4 crd;

  real domain_fac = corner.w;

  #ifndef EXACT_KEY
     crd.x = (int)roundf(__fdividef((pos.x - corner.x), domain_fac));
     crd.y = (int)roundf(__fdividef((pos.y - corner.y) , domain_fac));
     crd.z = (int)roundf(__fdividef((pos.z - corner.z) , domain_fac));
  #else
     crd.x = (int)((pos.x - corner.x) / domain_fac);
     crd.y = (int)((pos.y - corner.y) / domain_fac);
     crd.z = (int)((pos.z - corner.z) / domain_fac);
  #endif
    // uint2 key =  get_key_morton(crd);
    // body_key[idx] = make_uint4(key.x, key.y, 0,0);
    body_key[idx] = get_key(crd); //has to be PH key in order to prevent the need for sorting

}//end copyNode2grp

#endif

